#include <iostream>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <opencv2/opencv.hpp>
#include "utils.cuh"

void cudnn_conv2d(const Tensor& x_gpu, const Tensor& w_gpu, const Conv2dParam& conv_param, Tensor& y_gpu) {
    hipdnnHandle_t h_handle;
    CHECK_CUDNN(hipdnnCreate(&h_handle));

    hipdnnTensorDescriptor_t x_desc, y_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&x_desc));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&y_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        x_desc,
        HIPDNN_TENSOR_NHWC,
        HIPDNN_DATA_FLOAT,
        x_gpu.n,
        x_gpu.c,
        x_gpu.h,
        x_gpu.w
    ));

    // kernel
    hipdnnFilterDescriptor_t w_desc;
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&w_desc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(
        w_desc,
        HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NHWC,
        w_gpu.n,
        w_gpu.c,
        w_gpu.h,
        w_gpu.w
    ));

    // conv
    hipdnnConvolutionDescriptor_t conv_desc;
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(
        conv_desc,
        conv_param.pad_h,
        conv_param.pad_w,
        conv_param.u,
        conv_param.v,
        conv_param.dilation_h,
        conv_param.dilation_w,
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT
    ));

    // output
    hipdnnGetConvolution2dForwardOutputDim(conv_desc, x_desc, w_desc, &y_gpu.n, &y_gpu.c, &y_gpu.h, &y_gpu.w);
    y_gpu.alloc_gpu();

    // conv algorithm
    // hipdnnConvolutionFwdAlgoPerf_t algo_perf;
    hipdnnConvolutionFwdAlgo_t conv_algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;

    float alpha = 1.0f;
    float beta = 1.0f;
    float* out_ptr = y_gpu.get_ptr();
    CHECK_CUDNN(hipdnnConvolutionForward(
        h_handle,
        &alpha,
        x_desc,
        x_gpu.get_ptr(),
        w_desc,
        w_gpu.get_ptr(),
        conv_desc,
        conv_algo,
        nullptr,
        0,
        &beta,
        y_desc,
        out_ptr
    ));

    hipDeviceSynchronize();
    hipdnnDestroyTensorDescriptor(x_desc);
    hipdnnDestroyTensorDescriptor(y_desc);
    hipdnnDestroyFilterDescriptor(w_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(h_handle);
}


int main() {

    cv::Mat src = cv::imread("/home/penghuiwei/MyWorkspace/ubuntu/CPP/cudaCudnnConv/image/lena.jpg");
    cv::Mat src_fp;
    src.convertTo(src_fp, CV_32FC3);

    // Input
    Tensor tensor_x, tensor_w, tensor_y;
    tensor_x.alloc_gpu(1, 3, src.rows, src.cols);
    float* dev_ptr = tensor_x.get_ptr();
    CHECK_CUDA(hipMemcpy(dev_ptr, src_fp.data, tensor_x.size_byte, hipMemcpyHostToDevice));

    // kernel & conv
    Conv2dParam param;
    param.pad_h = param.pad_w = 1;
    param.dilation_h = param.dilation_w = 1;
    param.u = param.v = 1;
    make_kernel(tensor_w);

    cudnn_conv2d(tensor_x, tensor_w, param, tensor_y);

    cv::Mat dst_fp(cv::Size2d(tensor_y.w, tensor_y.h), CV_32FC(tensor_y.c));
    cv::Mat dst;
    CHECK_CUDA(hipMemcpy(dst_fp.data, tensor_y.get_ptr(), tensor_y.size_byte, hipMemcpyDeviceToHost));
    dst_fp.convertTo(dst, CV_8UC(tensor_y.c));

    cv::imwrite("dst.png", dst);

    // cv::imshow("src", src);
    // cv::imshow("dst", dst);
    // cv::waitKey(0);
}