#include <iostream>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <opencv2/opencv.hpp>
#include "utils.cuh"

using algo_perf_t = hipdnnConvolutionFwdAlgoPerf_t;

// if exit, algo_arr[0] will be best candidate
bool get_valid_best_algo(std::vector<algo_perf_t>& algo_arr) {
    auto it = std::remove_if(algo_arr.begin(), algo_arr.end(), [](algo_perf_t algo_perf){
        return algo_perf.status != HIPDNN_STATUS_SUCCESS;
    });
    algo_arr.erase(it, algo_arr.end());
    if(algo_arr.size() == 0) {
        std::runtime_error("Found no valid conv algorithm!");
    } 
    std::sort(algo_arr.begin(), algo_arr.end(), [](algo_perf_t algo1, algo_perf_t algo2){
        return algo1.time < algo2.time;
    });
    return algo_arr.size()>0;
}

void cudnn_conv2d(const Tensor& x_gpu, const Tensor& w_gpu, const Conv2dParam& conv_param, Tensor& y_gpu) {
    hipdnnHandle_t h_handle;
    CHECK_CUDNN(hipdnnCreate(&h_handle));

    hipdnnTensorDescriptor_t x_desc, y_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&x_desc));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&y_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        x_desc,
        HIPDNN_TENSOR_NHWC,
        HIPDNN_DATA_FLOAT,
        x_gpu.n,
        x_gpu.c,
        x_gpu.h,
        x_gpu.w
    ));

    // kernel
    hipdnnFilterDescriptor_t w_desc;
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&w_desc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(
        w_desc,
        HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NHWC,
        w_gpu.n,
        w_gpu.c,
        w_gpu.h,
        w_gpu.w
    ));

    // conv
    hipdnnConvolutionDescriptor_t conv_desc;
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(
        conv_desc,
        conv_param.pad_h,
        conv_param.pad_w,
        conv_param.u,
        conv_param.v,
        conv_param.dilation_h,
        conv_param.dilation_w,
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT
    ));

    // output
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(
        conv_desc, 
        x_desc, 
        w_desc, 
        &y_gpu.n, 
        &y_gpu.c, 
        &y_gpu.h, 
        &y_gpu.w
    ));
    y_gpu.alloc_gpu();
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        y_desc,
        HIPDNN_TENSOR_NHWC,
        HIPDNN_DATA_FLOAT,
        y_gpu.n,
        y_gpu.c,
        y_gpu.h,
        y_gpu.w
    ));

    // conv algorithm
    std::vector<algo_perf_t> algo_perf_arr;
    int request_cnt = 0;
    CHECK_CUDNN(cudnnGetConvolutionForwardAlgorithmMaxCount(h_handle, &request_cnt));
    algo_perf_arr.resize(request_cnt);
    int algo_count = 0;

    CHECK_CUDNN(hipdnnSetConvolutionMathType(conv_desc, CUDNN_FMA_MATH));

     // cudnnGetConvolutionForwardAlgorithm_v7
    CHECK_CUDNN(cudnnGetConvolutionForwardAlgorithm_v7(
        h_handle, 
        x_desc, 
        w_desc, 
        conv_desc, 
        y_desc,
        request_cnt,
        &algo_count,
        algo_perf_arr.data()
        ));

    if(!get_valid_best_algo(algo_perf_arr)) {
        std::runtime_error("Found no valid conv algorithm!");
    }
    hipdnnConvolutionFwdAlgo_t best_algo = algo_perf_arr[0].algo;

    size_t ws = 0;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
        h_handle, 
        x_desc, 
        w_desc, 
        conv_desc, 
        y_desc, 
        best_algo,
        &ws));
    void* workspace = nullptr;
    if(ws > 0) {
        CHECK_CUDA(hipMalloc(&workspace, ws));
    }

    // Forward
    float alpha = 1.0f;
    float beta = 1.0f;
    float* out_ptr = y_gpu.get_ptr();
    CHECK_CUDNN(hipdnnConvolutionForward(
        h_handle,
        &alpha,
        x_desc,
        x_gpu.get_ptr(),
        w_desc,
        w_gpu.get_ptr(),
        conv_desc,
        best_algo,
        workspace,
        ws,
        &beta,
        y_desc,
        out_ptr
    ));

    hipDeviceSynchronize();
    hipdnnDestroyTensorDescriptor(x_desc);
    hipdnnDestroyTensorDescriptor(y_desc);
    hipdnnDestroyFilterDescriptor(w_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(h_handle);
}


int main() {

    cv::Mat src = cv::imread("/home/penghuiwei/MyWorkspace/ubuntu/CPP/cudaCudnnConv/image/lena.jpg");
    cv::Mat src_fp;
    src.convertTo(src_fp, CV_32FC3);

    // Input
    Tensor tensor_x, tensor_w, tensor_y;
    tensor_x.alloc_gpu(1, 3, src.rows, src.cols);
    float* dev_ptr = tensor_x.get_ptr();
    CHECK_CUDA(hipMemcpy(dev_ptr, src_fp.data, tensor_x.size_byte, hipMemcpyHostToDevice));

    // kernel & conv
    Conv2dParam param;
    param.pad_h = param.pad_w = 1;
    param.dilation_h = param.dilation_w = 1;
    param.u = param.v = 1;
    make_kernel(tensor_w);

    cudnn_conv2d(tensor_x, tensor_w, param, tensor_y);

    cv::Mat dst_fp(cv::Size2d(tensor_y.w, tensor_y.h), CV_32FC(tensor_y.c));
    cv::Mat dst;
    CHECK_CUDA(hipMemcpy(dst_fp.data, tensor_y.get_ptr(), tensor_y.size_byte, hipMemcpyDeviceToHost));
    dst_fp.convertTo(dst, CV_8UC(tensor_y.c));

    cv::imwrite("dst.png", dst);

    // cv::imshow("src", src);
    // cv::imshow("dst", dst);
    // cv::waitKey(0);
}