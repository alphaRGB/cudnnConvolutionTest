#include <iostream>
#include <hipblas.h>
#include "utils.cuh"

#define CHECK_CUBLAS(status) \
    if (status != HIPBLAS_STATUS_SUCCESS) {  \
        std::cout << cublasGetStatusString(status) << std::endl; \
    }

// https://docs.nvidia.com/cuda/cublas/index.html
// https://zhuanlan.zhihu.com/p/441576790
void cublas_matmul(const Tensor& A, const Tensor& B, Tensor& C) {
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    // hipblasOperation_t 
    // hipblasSgemm(handle,)
}